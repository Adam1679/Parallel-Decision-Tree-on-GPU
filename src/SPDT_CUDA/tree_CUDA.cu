#include "hip/hip_runtime.h"

#include "array_CUDA.cu_inl"
#include "../SPDT_general/timing.h"
#include "panel.h"
#include <assert.h>
#include <queue>
#include <stdio.h>
#include <algorithm>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "array_CUDA.h"
#include "tree_CUDA.h"
#include "parser_CUDA.h"

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

float COMPRESS_TIME = 0.f;
float SPLIT_TIME = 0.f;
float COMMUNICATION_TIME = 0.f;
long long SIZE = 0 ;

int num_of_features = -1;
int num_of_classes = -1;
int max_bin_size = -1;
int max_num_leaves = -1;

__constant__ GlobalConstants cuConstTreeParams;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
 * Similar to update
 */
__global__ void
histogram_update_kernel() {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int data_id = blockIdx.x;
    int feature_id = threadIdx.x;
    int data_size = cuConstTreeParams.num_of_data;
    int num_of_features = cuConstTreeParams.num_of_features;

    if (data_id >= data_size || feature_id >= num_of_features)
       return;

    int* cuda_label_ptr = cuConstTreeParams.cuda_label_ptr;
    float* cuda_value_ptr = cuConstTreeParams.cuda_value_ptr;
    int* cuda_histogram_id_ptr = cuConstTreeParams.cuda_histogram_id_ptr;
	int num_of_classes = cuConstTreeParams.num_of_classes;
    int max_bin_size = cuConstTreeParams.max_bin_size;
    float* _histogram_ = cuConstTreeParams.cuda_histogram_ptr;
    CUDA_update_array(
        cuda_histogram_id_ptr[data_id], 
        feature_id, 
        cuda_label_ptr[data_id], 
        cuda_value_ptr[data_id * num_of_features + feature_id],
        num_of_features,
        num_of_classes,
        max_bin_size,
        _histogram_);
}

// block_num: size of unlabeled leaves
// thread_num: num_of_features
__global__ void
histogram_update_kernel_2() {
    int histogram_id = blockIdx.x;
    int feature_id = threadIdx.x;    
    int num_of_features = cuConstTreeParams.num_of_features;    

    if (histogram_id >= cuConstTreeParams.max_num_leaves || feature_id >= num_of_features)
       return;

    int* cuda_label_ptr = cuConstTreeParams.cuda_label_ptr;
    float* cuda_value_ptr = cuConstTreeParams.cuda_value_ptr;
    int* cuda_histogram_id_ptr = cuConstTreeParams.cuda_histogram_id_ptr;
	int num_of_classes = cuConstTreeParams.num_of_classes;
    int max_bin_size = cuConstTreeParams.max_bin_size;
    float* _histogram_ = cuConstTreeParams.cuda_histogram_ptr;

    for (int i = 0; i < cuConstTreeParams.num_of_data; i++) {
        if (cuda_histogram_id_ptr[i] != histogram_id) continue;
        CUDA_update_array(
            cuda_histogram_id_ptr[i], 
            feature_id, 
            cuda_label_ptr[i], 
            cuda_value_ptr[i * num_of_features + feature_id],
            num_of_features,
            num_of_classes,
            max_bin_size,
            _histogram_);
        __syncthreads();
    }
    
}

SplitPoint::SplitPoint()
{
    feature_id = -1;
    feature_value = 0;
    entropy = 0;
}

SplitPoint::SplitPoint(int feature_id, float feature_value)
{
    this->feature_id = feature_id;
    this->feature_value = feature_value;
    this->entropy = 0;    
}
/*
 * Reture True if the data is larger or equal than the split value
 */
bool SplitPoint::decision_rule(int data_index, Dataset *datasetPointer)
{        
    dbg_ensures(entropy >= -EPS);    
    dbg_ensures(gain >= -EPS);    
    dbg_ensures(feature_id >= 0);
    dbg_ensures(feature_id < num_of_features);     
    assert(datasetPointer->value_ptr != NULL);        
    assert(data_index * num_of_features + feature_id >= 0); 

    // printf("data_index %d\n", data_index);
    // printf("num_of_features %d\n", num_of_features);
    // printf("feature_id %d\n", feature_id);
    // printf("datasetPointer->num_of_data %d\n", datasetPointer->num_of_data);

    assert((long long int) data_index * num_of_features + feature_id < 
        (long long int) datasetPointer->num_of_data * num_of_features);    
    bool result = datasetPointer->value_ptr[data_index * num_of_features + feature_id] >= feature_value;        
    return result;
}

// constructor function
TreeNode::TreeNode(int depth, int id, Dataset* datasetPointer)
{
    this->id = id;
    this->depth = depth;
    is_leaf = false;
    label = -1;    
    histogram_id = -1;    
    left_node = NULL;
    right_node = NULL;
    entropy = -1.f;
    num_pos_label=0;
    data_size = 0;
    is_leaf = true;
    this->datasetPointer = datasetPointer;
}


void TreeNode::init()
{
    label = -1;
    histogram_id = -1;    
    left_node = NULL;
    right_node = NULL;
    is_leaf = true;
    return;
}

/*
 * Set label for the node as the majority class.
 */
void TreeNode::set_label()
{
    this->is_leaf = true;
    this->label = (this->num_pos_label >= (int)this->data_size / 2) ? POS_LABEL : NEG_LABEL;
}

/*
 * This function split the data according to the best split feature id and value.
 * The data would be appended to the `left` if the data value is smaller than the split value
 */
void TreeNode::split(SplitPoint &best_split, TreeNode* left, TreeNode* right)
{    
    assert(left != NULL);
    assert(right != NULL);
    this->split_ptr = best_split;
    this->entropy = best_split.entropy;
    int num_pos_label_left=0;
    int num_pos_label_right=0;
    for (int i = 0; i < this->data_ptr.size(); i++) {        
        int data_index = this->data_ptr[i];
        if (best_split.decision_rule(data_index, this->datasetPointer)) {                        
            right->data_ptr.push_back(data_index);
            this->datasetPointer->histogram_id_ptr[data_index] = right->id;
            assert(this->datasetPointer->label_ptr != NULL);
            num_pos_label_right = (this->datasetPointer->label_ptr[data_index] == POS_LABEL) ? num_pos_label_right + 1 : num_pos_label_right;
            right->data_size++;
        } else {            
            left->data_ptr.push_back(data_index);
            this->datasetPointer->histogram_id_ptr[data_index] = left->id;
            assert(this->datasetPointer->label_ptr != NULL);           
            num_pos_label_left = (this->datasetPointer->label_ptr[data_index] == POS_LABEL) ? num_pos_label_left + 1 : num_pos_label_left;
            left->data_size++;
        }
    }
   
    left->num_pos_label = num_pos_label_left;
    right->num_pos_label = num_pos_label_right;

    dbg_assert(left->num_pos_label >= 0);
    dbg_assert(right->num_pos_label >= 0);
    dbg_assert(left->num_pos_label + right->num_pos_label == this->num_pos_label);
    dbg_assert(left->data_size + right->data_size == this->data_size);    
}

void TreeNode::printspaces() {
    int i = 0;
    for (i = 0; i < depth * 2; i++) {
        printf(" ");
    }
}

void TreeNode::print() {
    printspaces();
    printf("TreeNode: \n");
    printspaces();
    printf("depth: %d\n", depth);
    printspaces();
    printf("label %d\n", label);
    printspaces();
    printf("is_leaf %d\n", is_leaf);
    printspaces();
    printf("hasLeft: %d\n", left_node != NULL);
    printspaces();
    printf("hasRight: %d\n", right_node != NULL);    
    if (left_node != NULL) {
        left_node->print();
    }

    if (right_node != NULL) {
        right_node->print();
    }
}

void TreeNode::clear(){
    if (left_node != NULL) left_node->clear();
    if (right_node != NULL) right_node->clear();    
}

DecisionTree::DecisionTree()
{
    this->max_num_leaves = 64;
    this->max_depth = -1;
    this->min_node_size = 1;
    this->depth = 0;
    this->num_leaves = 0;    
    this->cur_depth = 0;
    this->root = NULL;    
    this->min_gain = 1e-3;
    this->num_nodes = 0;

}

DecisionTree::~DecisionTree(){    
    root->clear();
}


void DecisionTree::initCUDA() {
    int data_size = this->datasetPointer->num_of_data;
    // Construct the histogram. and navigate each data to its leaf.  
    gpuErrchk(hipMalloc(&cuda_histogram_ptr, sizeof(float) * SIZE));
    gpuErrchk(hipMalloc(&cuda_label_ptr, sizeof(int) * data_size));
    gpuErrchk(hipMalloc(&cuda_value_ptr, sizeof(float) * data_size * num_of_features));
    gpuErrchk(hipMalloc(&cuda_histogram_id_ptr, sizeof(int) * data_size));
    gpuErrchk(hipMemcpy(cuda_histogram_ptr,
        histogram,
        sizeof(float) * SIZE,
        hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(cuda_label_ptr,
        this->datasetPointer->label_ptr,
        sizeof(int) * data_size,
        hipMemcpyHostToDevice)); 

    gpuErrchk(hipMemcpy(cuda_value_ptr,
        this->datasetPointer->value_ptr,
        sizeof(float) * data_size * num_of_features,
        hipMemcpyHostToDevice));  

    gpuErrchk(hipMemcpy(cuda_histogram_id_ptr,
        this->datasetPointer->histogram_id_ptr,
        sizeof(int) * data_size,
        hipMemcpyHostToDevice)); 

    GlobalConstants params;
    params.cuda_histogram_id_ptr = cuda_histogram_id_ptr;
    params.cuda_histogram_ptr = cuda_histogram_ptr;
    params.cuda_label_ptr = cuda_label_ptr;
    params.cuda_value_ptr = cuda_value_ptr;
    params.num_of_data = data_size;
    params.num_of_classes = num_of_classes;
    params.max_bin_size = max_bin_size;
    params.num_of_features = num_of_features;
    params.max_num_leaves = max_num_leaves;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuConstTreeParams), &params, sizeof(GlobalConstants)));

}

void DecisionTree::terminateCUDA() {
    hipFree(cuda_histogram_ptr);
    hipFree(cuda_label_ptr);
    hipFree(cuda_value_ptr);
    hipFree(cuda_histogram_id_ptr);
}

/* 
 * Return true if the node should be a leaf.
 * This is determined by the min-node-size, max-depth, max_num_leaves
*/
bool DecisionTree::is_terminated(TreeNode *node)
{
    if (min_node_size != -1 && node->data_size <= min_node_size)
    {
        printf("Node [%d] terminated: min_node_size=%d >= %d\n", node->id, min_node_size, node->data_size);
        return true;
    }

    if (max_depth != -1 && node->depth >= this->max_depth)
    {
        printf("Node [%d] terminated: max_depth\n", node->id);
        return true;
    }

    if (max_num_leaves != -1 && this->num_leaves >= this->max_num_leaves)
    {
        printf("Node [%d] terminated: max_num_leaves\n", node->id);
        return true;
    }

    if (!node->num_pos_label || node->num_pos_label == (int) node->data_size){
        dbg_assert(node->entropy < EPS);
        printf("Node [%d] terminated: all samples belong to same class\n",node->id);
        return true; 
    }
    printf("[%d] num_data=%d, num_pos=%d\n", node->id, node->data_size, node->num_pos_label);
    return false;
}

void DecisionTree::initialize(Dataset &train_data, const int batch_size){
    this->datasetPointer = &train_data;    
    root = new TreeNode(0, this->num_nodes++, datasetPointer);  
    root->data_size = train_data.num_of_data;
    for (int i = 0; i < root->data_size; i++) {
        root->data_ptr.push_back(i);
    }

    if (histogram != NULL) {        
        delete[] histogram;
    }    
    SIZE = (long long) max_num_leaves * num_of_features * num_of_classes * ((max_bin_size + 1) * 2 + 1);    
    printf("Init Root Node [%.4f] MB\n", SIZE * sizeof(float) / 1024.f / 1024.f);
    
    histogram = new float[SIZE];
    memset(histogram, 0, SIZE * sizeof(float));  
    printf("Init success\n");
}

void DecisionTree::train(Dataset &train_data, const int batch_size)
{
    int hasNext = TRUE;
    initialize(train_data, batch_size);
	while (TRUE) {
		hasNext = train_data.streaming_read_data(batch_size);	
        printf("Train size (%d, %d, %d)\n", train_data.num_of_data, 
                num_of_features, num_of_classes);
                
        Timer t = Timer();
        t.reset();
        initCUDA();
        COMMUNICATION_TIME += t.elapsed();

        train_on_batch(train_data);        
		if (!hasNext) break;
	}		
    
    train_data.close_read_data();
    terminateCUDA(); 
    printf("COMPRESS TIME: %f\nSPLIT TIME: %f\nCOMMUNICATION TIME: %f\n", 
        COMPRESS_TIME, SPLIT_TIME, COMMUNICATION_TIME);   
    return;
}

float DecisionTree::test(Dataset &test_data) {    

    int i = 0;
    int correct_num = 0;
    test_data.streaming_read_data(test_data.num_of_data);   
    this->datasetPointer = &test_data;     

    for (i = 0; i < test_data.num_of_data; i++) {        
        assert(navigate(i, &test_data)->label != -1);        
        if (navigate(i, &test_data)->label == test_data.label_ptr[i]) {
            correct_num++;
        }
    }    
    return (float)correct_num / (float)test_data.num_of_data;
}

/*
 * Calculate the entropy gain = H(Y) - H(Y|X)
 * H(Y|X) needs parameters p(X<a), p(Y=0|X<a), p(Y=0|X>=a)
 * Assuming binary classification problem
 */
void get_gain(TreeNode* node, SplitPoint& split, int feature_id){
    int total_sum = node->data_size;
    dbg_ensures(total_sum > 0);
    float sum_class_0 = get_total_array(node->histogram_id, feature_id, NEG_LABEL);
    float sum_class_1 = get_total_array(node->histogram_id, feature_id, POS_LABEL);
    printf("(int)sum_class_1: %d\n", (int)sum_class_1);
    printf("node->num_pos_label: %d\n", node->num_pos_label);    
    dbg_assert((int)sum_class_1 == node->num_pos_label);
    float left_sum_class_0 = sum_array(node->histogram_id, feature_id, NEG_LABEL, split.feature_value);
    float right_sum_class_0 = sum_class_0 - left_sum_class_0;
    float left_sum_class_1 = sum_array(node->histogram_id, feature_id, POS_LABEL, split.feature_value);
    float right_sum_class_1 = sum_class_1 - left_sum_class_1;
    float left_sum = left_sum_class_0 + left_sum_class_1;
    float right_sum = right_sum_class_0 + right_sum_class_1;

    float px = (left_sum_class_0 + left_sum_class_1) / (1.0 * total_sum); // p(x<a)
    float py_x0 = (left_sum <= EPS) ? 0.f : left_sum_class_0 / left_sum;                            // p(y=0|x < a)
    float py_x1 = (right_sum <= EPS) ? 0.f : right_sum_class_0 / right_sum;                          // p(y=0|x >= a)
    // printf("sum_class_1=%f, sum_class_0=%f, right_sum = %f, right_sum_class_0 = %f right_sum_class_1= %f\n", sum_class_1, sum_class_0, right_sum, right_sum_class_0, right_sum_class_1);
    // printf("py_x0 = %f, py_x1 = %f\n", py_x0, py_x1);
    dbg_ensures(py_x0 >= -EPS && py_x0 <= 1+EPS);
    dbg_ensures(py_x1 >= -EPS && py_x1 <= 1+EPS);
    dbg_ensures(px >= -EPS && px <= 1+EPS);
    float entropy_left = ((1-py_x0) < EPS || py_x0 < EPS) ? 0 : -py_x0 * log2((double)py_x0) - (1-py_x0)*log2((double)1-py_x0);
    float entropy_right = ((1-py_x1) < EPS || py_x1 < EPS) ? 0 : -py_x1 * log2((double)py_x1) - (1-py_x1)*log2((double)1-py_x1);
    float H_YX = px * entropy_left + (1-px) * entropy_right;
    float px_prior = sum_class_0 / (sum_class_0 + sum_class_1);
    dbg_ensures(px_prior > 0 && px_prior < 1);
    split.entropy = ((1-px_prior) < EPS || px_prior < EPS) ? 0 : -px_prior * log2((double)px_prior) - (1-px_prior) * log2((double)1-px_prior);
    split.gain = split.entropy - H_YX;
    // printf("%f = %f - %f\n", split.gain, split.entropy, H_YX);
    dbg_ensures(split.gain + EPS >= 0);
}

/*
 * This function return the best split point at a given leaf node.
 * Best split is store in `split`
*/
void DecisionTree::find_best_split(TreeNode *node, SplitPoint &split)
{              
    assert(node != NULL);
    
    float* buf_merge = new float[2 * max_bin_size + 1];
    SplitPoint best_split = SplitPoint();    
    for (int i = 0; i < num_of_features; i++)
    {
        // merge different labels
        // put the result back into (node->histogram_id, i, 0)
        float* histo_for_class_0 = get_histogram_array(node->histogram_id, i, NEG_LABEL);
        float* histo_for_class_1 = get_histogram_array(node->histogram_id, i, POS_LABEL);
        // initialize the buf_merge
        memcpy(buf_merge, histo_for_class_0, sizeof(float) * (2 * max_bin_size + 1));
        cout << "histo_0: ";
        print_array(histo_for_class_0);
        cout << "histo_1: ";
        print_array(histo_for_class_1);
        std::vector<float> possible_splits;
        merge_array_pointers(buf_merge, histo_for_class_1);
        cout << "merged: ";
        print_array(buf_merge);
        uniform_array(possible_splits, node->histogram_id, i, 0, buf_merge);
        dbg_assert(possible_splits.size() <= max_bin_size);
        for (auto& split_value: possible_splits)
        {
            SplitPoint t = SplitPoint(i, split_value);
            get_gain(node, t, i);
            if (best_split.gain < t.gain)
                best_split = t;
        }
    }
    split = best_split;
    delete[] buf_merge;
}

/*
 * This function compress the data into histograms.
 * Each unlabeled leaf would have a (num_feature, num_class) histograms
 * This function takes the assumption that each leaf is re-initialized (we use a batch mode)
*/
void DecisionTree::compress(vector<TreeNode *> &unlabeled_leaf) {
    // int block_num = this->datasetPointer->num_of_data;
    // int thread_per_block = num_of_features;                                 
    // histogram_update_kernel<<<block_num, thread_per_block>>>();      
        
    int block_num = unlabeled_leaf.size();
    int thread_per_block = num_of_features;
    
    gpuErrchk(hipMemcpy(cuda_histogram_ptr,
        histogram,
        sizeof(float) * SIZE,
        hipMemcpyHostToDevice));

    histogram_update_kernel_2<<<block_num, thread_per_block>>>();         

    hipDeviceSynchronize();
    hipMemcpy(histogram,
        cuda_histogram_ptr,
        sizeof(float) * SIZE,
        hipMemcpyDeviceToHost);  

    // float *histo = NULL;
    // int bin_size = 0;
    // for (int i = 0; i < num_of_features; i++) {
    //     for (int j = 0; j < num_of_classes; j++) {
    //         histo = get_histogram_array(0, i, j);
    //         bin_size = get_bin_size(histo);
    //         printf("[%d][%d]: bin_size %d\n", i, j, bin_size);
    //     }
    // }    
    
    // // sequential version for DEBUG!!!
    // // Construct the histogram. and navigate each data to its leaf.    
    // for (int data_id = 0; data_id < this->datasetPointer->num_of_data; data_id++) {
    //     for (int feature_id = 0; feature_id < num_of_features; feature_id++) {
    //         update_array(
    //             this->datasetPointer->histogram_id_ptr[data_id], 
    //             feature_id, 
    //             this->datasetPointer->label_ptr[data_id], 
    //             this->datasetPointer->value_ptr[data_id * num_of_features + feature_id]);
    //     }        
    // }   
}

/*
 * Serial version of training.
*/
void DecisionTree::train_on_batch(Dataset &train_data)
{       
    float pos_rate = ((float) train_data.num_pos_label) / train_data.num_of_data;
    dbg_assert(pos_rate > 0 && pos_rate < 1);
    root->num_pos_label = train_data.num_pos_label;
    root->entropy = - pos_rate * log2((double)pos_rate) - (1-pos_rate) * log2((double)(1-pos_rate));
    batch_initialize(root); // Reinitialize every leaf in T as unlabeled.
    vector<TreeNode *> unlabeled_leaf = __get_unlabeled(root);
    dbg_assert(unlabeled_leaf.size() <= max_num_leaves);
    while (!unlabeled_leaf.empty())
    {        
        // each while loop would add a new level node.
        this->cur_depth++;
        printf("Depth [%d] finished\n", this->cur_depth);
        vector<TreeNode *> unlabeled_leaf_new; 
        if (unlabeled_leaf.size() > max_num_leaves) {
            for (int i = 0; i < unlabeled_leaf.size(); i++) {
                unlabeled_leaf[i]->set_label();
                this->num_leaves++;
            }
            break;
        }       
        init_histogram(unlabeled_leaf);
        Timer t1 = Timer();
        t1.reset();
        compress(unlabeled_leaf); 
        COMPRESS_TIME += t1.elapsed();         
        for (auto &cur_leaf : unlabeled_leaf)
        {            
            if (is_terminated(cur_leaf))
            {         
                cur_leaf->set_label();
                this->num_leaves++;             
            }
            else
            {                
                SplitPoint best_split;                
                Timer t2 = Timer();
                t2.reset();
                find_best_split(cur_leaf, best_split);
                SPLIT_TIME += t2.elapsed();                
                dbg_ensures(best_split.gain >= -EPS);
                if (best_split.gain <= min_gain){
                    printf("Node terminated: gain=%.4f <= %.4f\n", min_node_size, best_split.gain, min_gain);
                    cur_leaf->set_label();
                    this->num_leaves++;               
                    continue;
                }
                cur_leaf->left_node = new TreeNode(this->cur_depth, this->num_nodes++, datasetPointer);
                cur_leaf->right_node = new TreeNode(this->cur_depth, this->num_nodes++, datasetPointer);
                cur_leaf->split(best_split, cur_leaf->left_node, cur_leaf->right_node);
                cur_leaf->is_leaf = false;
                cur_leaf->label = -1;
                unlabeled_leaf_new.push_back(cur_leaf->left_node);
                unlabeled_leaf_new.push_back(cur_leaf->right_node);
            }
        }
        unlabeled_leaf = unlabeled_leaf_new;
        unlabeled_leaf_new.clear(); 
    }
    self_check();    
}

void DecisionTree::self_check(){
    queue<TreeNode *> q;
    q.push(root);
    int count_leaf=0;
    int count_nodes=0;
    while (!q.empty())
    {
        auto tmp_ptr = q.front();
        q.pop();
        count_nodes++;
        if (tmp_ptr == NULL)
        {
            // should never reach here.
            fprintf(stderr, "ERROR: The tree contains node that have only one child\n");
            exit(-1);
        }
        else if ((tmp_ptr->left_node == NULL) && (tmp_ptr->right_node == NULL))
        {
            dbg_requires(tmp_ptr->is_leaf);
            dbg_requires(tmp_ptr->label == POS_LABEL || tmp_ptr->label == NEG_LABEL);
            count_leaf++;
        }
        else
        {
            dbg_requires(!tmp_ptr->is_leaf);
            dbg_requires(tmp_ptr->label == -1);
            q.push(tmp_ptr->left_node);
            q.push(tmp_ptr->right_node);
        }
    }
    dbg_assert(count_leaf == num_leaves);
    dbg_assert(count_nodes == num_nodes);
    printf("------------------------------------------------\n");
    printf("| Num_leaf: %d, num_nodes: %d, max_depth: %d | \n", num_leaves, num_nodes, cur_depth);
    printf("------------------------------------------------\n");

}


/* 
 * This function reture all the unlabeled leaf nodes in a breadth-first manner.
*/
vector<TreeNode *> DecisionTree::__get_unlabeled(TreeNode *node)
{
    queue<TreeNode *> q;
    q.push(node);
    vector<TreeNode *> ret;
    while (!q.empty())
    {
        auto tmp_ptr = q.front();
        q.pop();
        if (tmp_ptr == NULL)
        {
            // should never reach here.
            fprintf(stderr, "ERROR: The tree contains node that have only one child\n");
            exit(-1);
        }
        else if ((tmp_ptr->left_node == NULL) && (tmp_ptr->right_node == NULL) && tmp_ptr->label < 0)
        {
            ret.push_back(tmp_ptr);
        }
        else
        {
            q.push(tmp_ptr->left_node);
            q.push(tmp_ptr->right_node);
        }
    }
    return ret;
}

/*
 * initialize each leaf as unlabeled.
 */
void DecisionTree::batch_initialize(TreeNode *node)
{
    if (node == NULL)
    {
        // should never reach here.
        fprintf(stderr, "ERROR: The tree contains node that have only one child\n");
        exit(-1);
    }
    else if ((node->left_node == NULL) && (node->right_node == NULL))
    {
       node->init();
    }
    else
    {
        batch_initialize(node->left_node);
        batch_initialize(node->right_node);
    }
    return;
}


/*
 *
 */
TreeNode *DecisionTree::navigate(int data_index, Dataset *datasetPointer)
{
    TreeNode *ptr = this->root;
    while (!ptr->is_leaf)
    {
        dbg_assert(ptr->right_node != NULL && ptr->left_node != NULL);
        ptr = (ptr->split_ptr.decision_rule(data_index, datasetPointer)) ? ptr->right_node : ptr->left_node;
    }
    return ptr;
}

__global__ void
navigate_sample_kernel(int unlabeled_leaf_size, int *cuda_histogram_id_2_node_id) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;    
    int data_size = cuConstTreeParams.num_of_data;
    
    if (index >= data_size) return;
    int* cuda_histogram_id_ptr = cuConstTreeParams.cuda_histogram_id_ptr;
	
    for (int i = 0; i < unlabeled_leaf_size; i++) {
        if (cuda_histogram_id_ptr[index] == cuda_histogram_id_2_node_id[i]) {
            cuda_histogram_id_ptr[index] = i;
            break;
        }
    }    
    __syncthreads(); 
}

/*
 * This function initialize the histogram for each unlabeled leaf node.
 * Also, potentially, it would free the previous histogram.
 */
void DecisionTree::init_histogram(vector<TreeNode *> &unlabeled_leaf)
{
    int c = 0;      
    assert(unlabeled_leaf.size() <= max_num_leaves);   
    
    // map the histogram id to node id
    // TODO: use another map? map node id to histogram id
    int histogram_id_2_node_id[unlabeled_leaf.size()];
    int* cuda_histogram_id_2_node_id;
    
    for (auto &p : unlabeled_leaf) {
        p->histogram_id = c++;   
        // build an array index, between node's node_id and histogram_id
        histogram_id_2_node_id[p->histogram_id] = p->id;     
    }   
        
    int thread_num = 128;
    int block_num = (this->datasetPointer->num_of_data + thread_num - 1) / thread_num;        

    // previously, store the node id in histogram_id_ptr
    gpuErrchk(hipMemcpy(cuda_histogram_id_ptr,
        this->datasetPointer->histogram_id_ptr,
        sizeof(int) * this->datasetPointer->num_of_data,
        hipMemcpyHostToDevice)); 

    gpuErrchk(hipMalloc(&cuda_histogram_id_2_node_id, sizeof(int) * unlabeled_leaf.size()));

    gpuErrchk(hipMemcpy(cuda_histogram_id_2_node_id,
        histogram_id_2_node_id,
        sizeof(int) * unlabeled_leaf.size(),
        hipMemcpyHostToDevice)); 

    // change the node id into histogram id
    navigate_sample_kernel<<<block_num, thread_num>>>(unlabeled_leaf.size(), cuda_histogram_id_2_node_id); 
    hipDeviceSynchronize();  

    memset(histogram, 0, sizeof(float) * SIZE);       
}
