#include "hip/hip_runtime.h"
#include "tree_CUDA.h"
#include "parser_CUDA.h"
#include "array_CUDA.h"
#include "../SPDT_general/timing.h"
#include "panel.h"
#include <assert.h>
#include <queue>
#include <stdio.h>
#include <algorithm>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

float COMPRESS_TIME = 0.f;
float SPLIT_TIME = 0.f;
float COMMUNICATION_TIME = 0.f;
long long SIZE = 0 ;

int num_of_features = -1;
int num_of_classes = -1;
int max_bin_size = -1;
int max_num_leaves = -1;

__global__ void
navigate_samples_kernel() {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void
histogram_update_kernel(
    int data_size, 
    int num_of_threads,    
    float *cuda_histogram_ptr, 
    int *cuda_label_ptr,
    float *cuda_value_ptr,
    int *cuda_histogram_id_ptr,
    int num_of_features,
    int num_of_classes,
    int max_bin_size) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // blockIdx.x: the update data id
    // threadIdx.x: the update feature id

    if (blockIdx.x >= data_size || threadIdx.x >= num_of_threads)
       return;
    
    // re-write part of the DecisionTree::compress function      
    update_array(
        cuda_histogram_id_ptr[blockIdx.x], 
        threadIdx.x, 
        cuda_label_ptr[blockIdx.x], 
        cuda_value_ptr[blockIdx.x * num_of_features + threadIdx.x],
        cuda_histogram_ptr,
        num_of_features,
        num_of_classes,
        max_bin_size);
}

SplitPoint::SplitPoint()
{
    feature_id = -1;
    feature_value = 0;
    entropy = 0;
}

SplitPoint::SplitPoint(int feature_id, float feature_value, Dataset* datasetPointer)
{
    this->feature_id = feature_id;
    this->feature_value = feature_value;
    this->entropy = 0;
    this->datasetPointer = datasetPointer;
}
/*
 * Reture True if the data is larger or equal than the split value
 */
bool SplitPoint::decision_rule(int data_index)
{
    dbg_ensures(entropy >= -EPS);
    dbg_ensures(gain >= -EPS);
    dbg_ensures(feature_id >= 0);
    return datasetPointer->value_ptr[data_index * num_of_features + feature_id] >= feature_value;    
}

// constructor function
TreeNode::TreeNode(int depth, int id, Dataset* datasetPointer)
{
    this->id = id;
    this->depth = depth;
    is_leaf = false;
    label = -1;    
    histogram_id = -1;    
    left_node = NULL;
    right_node = NULL;
    entropy = -1.f;
    num_pos_label=0;
    data_size = 0;
    is_leaf = true;
    this->datasetPointer = datasetPointer;
}


void TreeNode::init()
{
    label = -1;
    histogram_id = -1;    
    left_node = NULL;
    right_node = NULL;
    is_leaf = true;
    return;
}

/*
 * Set label for the node as the majority class.
 */
void TreeNode::set_label()
{
    this->is_leaf = true;
    this->label = (this->num_pos_label >= (int)this->data_size / 2) ? POS_LABEL : NEG_LABEL;
}

/*
 * This function split the data according to the best split feature id and value.
 * The data would be appended to the `left` if the data value is smaller than the split value
 */
void TreeNode::split(SplitPoint &best_split, TreeNode* left, TreeNode* right)
{
    this->split_ptr = best_split;
    this->entropy = best_split.entropy;
    float split_value = best_split.feature_value;
    int num_pos_label_left=0;
    int num_pos_label_right=0;
    for (int i = 0; i < this->datasetPointer->num_of_data; i++) {
        if (this->datasetPointer->histogram_id_ptr[i] != this->histogram_id) {
            continue;
        }
        float p_value = this->datasetPointer->value_ptr[i * num_of_features + best_split.feature_id];
        if (best_split.decision_rule(i)) {
            this->datasetPointer->histogram_id_ptr[i] = right->histogram_id;
            num_pos_label_right = (this->datasetPointer->label_ptr[i] == POS_LABEL) ? num_pos_label_right + 1 : num_pos_label_right;
            right->data_size++;
        } else {
            this->datasetPointer->histogram_id_ptr[i] = left->histogram_id;
            num_pos_label_left = (this->datasetPointer->label_ptr[i] == POS_LABEL) ? num_pos_label_left + 1 : num_pos_label_left;
            left->data_size++;
        }
    }
   
    left->num_pos_label = num_pos_label_left;
    right->num_pos_label = num_pos_label_right;

    dbg_assert(left->num_pos_label >= 0);
    dbg_assert(right->num_pos_label >= 0);
    dbg_assert(left->num_pos_label + right->num_pos_label == this->num_pos_label);
}

void TreeNode::printspaces() {
    int i = 0;
    for (i = 0; i < depth * 2; i++) {
        printf(" ");
    }
}

void TreeNode::print() {
    printspaces();
    printf("TreeNode: \n");
    printspaces();
    printf("depth: %d\n", depth);
    printspaces();
    printf("label %d\n", label);
    printspaces();
    printf("is_leaf %d\n", is_leaf);
    printspaces();
    printf("hasLeft: %d\n", left_node != NULL);
    printspaces();
    printf("hasRight: %d\n", right_node != NULL);    
    if (left_node != NULL) {
        left_node->print();
    }

    if (right_node != NULL) {
        right_node->print();
    }
}

void TreeNode::clear(){
    if (left_node != NULL) left_node->clear();
    if (right_node != NULL) right_node->clear();    
}

DecisionTree::DecisionTree()
{
    this->max_num_leaves = 64;
    this->max_depth = -1;
    this->min_node_size = 1;
    this->depth = 0;
    this->num_leaves = 0;    
    this->cur_depth = 0;
    this->root = NULL;    
    this->min_gain = 1e-3;
    this->num_nodes = 0;

}

DecisionTree::~DecisionTree(){    
    root->clear();
}

void DecisionTree::initCUDA() {
    int data_size = this->datasetPointer->num_of_data;
    // Construct the histogram. and navigate each data to its leaf.  
    long long number = (long long) max_num_leaves * num_of_features * num_of_classes * ((max_bin_size + 1) * 2 + 1);        

    hipMalloc((void **)&cuda_histogram_ptr, sizeof(float) * number);
    hipMalloc((void **)&cuda_label_ptr, sizeof(int) * data_size);
    hipMalloc((void **)&cuda_value_ptr, sizeof(float) * data_size * num_of_features);
    hipMalloc((void **)&cuda_histogram_id_ptr, sizeof(int) * data_size);

    hipMemcpy(cuda_histogram_ptr,
        histogram,
        sizeof(float) * number,
        hipMemcpyHostToDevice);
    hipMemcpy(cuda_label_ptr,
        this->datasetPointer->label_ptr,
        sizeof(int) * data_size,
        hipMemcpyHostToDevice);  
    hipMemcpy(cuda_value_ptr,
        this->datasetPointer->value_ptr,
        sizeof(float) * data_size * num_of_features,
        hipMemcpyHostToDevice);  
    hipMemcpy(cuda_histogram_id_ptr,
        this->datasetPointer->histogram_id_ptr,
        sizeof(int) * data_size,
        hipMemcpyHostToDevice); 
}

void DecisionTree::terminateCUDA() {
    hipFree(cuda_histogram_ptr);
    hipFree(cuda_label_ptr);
    hipFree(cuda_value_ptr);
    hipFree(cuda_histogram_id_ptr);
}

/* 
 * Return true if the node should be a leaf.
 * This is determined by the min-node-size, max-depth, max_num_leaves
*/
bool DecisionTree::is_terminated(TreeNode *node)
{
    if (min_node_size != -1 && node->data_size <= min_node_size)
    {
        printf("Node [%d] terminated: min_node_size=%d >= %d\n", node->id, min_node_size, node->data_size);
        return true;
    }

    if (max_depth != -1 && node->depth >= this->max_depth)
    {
        printf("Node [%d] terminated: max_depth\n", node->id);
        return true;
    }

    if (max_num_leaves != -1 && this->num_leaves >= this->max_num_leaves)
    {
        printf("Node [%d] terminated: max_num_leaves\n", node->id);
        return true;
    }

    if (!node->num_pos_label || node->num_pos_label == (int) node->data_size){
        dbg_assert(node->entropy < EPS);
        printf("Node [%d] terminated: all samples belong to same class\n",node->id);
        return true; 
    }
    printf("[%d] num_data=%d, num_pos=%d\n", node->id, node->data_size, node->num_pos_label);
    return false;
}

void DecisionTree::initialize(Dataset &train_data, const int batch_size){
    this->datasetPointer = &train_data;    
    root = new TreeNode(0, this->num_nodes++, datasetPointer);  
    root->data_size = train_data.num_of_data;
    if (histogram != NULL) {        
        delete[] histogram;
    }
    
    SIZE  = (long long) max_num_leaves * num_of_features * num_of_classes * ((max_bin_size + 1) * 2 + 1);    
    printf("Init Root Node [%.4f] MB\n", SIZE * sizeof(float) / 1024.f / 1024.f);
    
    histogram = new float[SIZE];
    memset(histogram, 0, SIZE * sizeof(float));          
    printf("Init success\n");
}

void DecisionTree::train(Dataset &train_data, const int batch_size)
{
    int hasNext = TRUE;
    initialize(train_data, batch_size);
	while (TRUE) {
		hasNext = train_data.streaming_read_data(batch_size);	
        printf("Train size (%d, %d, %d)\n", train_data.num_of_data, 
                num_of_features, num_of_classes);
                
        Timer t = Timer();
        t.reset();
        initCUDA();
        COMMUNICATION_TIME += t.elapsed();

        train_on_batch(train_data);        
		if (!hasNext) break;
	}		
    
    train_data.close_read_data();
    terminateCUDA(); 
    printf("COMPRESS TIME: %f\nSPLIT TIME: %f\nCOMMUNICATION TIME: %f\n", 
        COMPRESS_TIME, SPLIT_TIME, COMMUNICATION_TIME);   
    return;
}

float DecisionTree::test(Dataset &test_data) {    

    int i = 0;
    int correct_num = 0;
    test_data.streaming_read_data(test_data.num_of_data);

    for (i = 0; i < test_data.num_of_data; i++) {
        assert(navigate(i)->label != -1);        
        if (navigate(i)->label == test_data.label_ptr[i]) {
            correct_num++;
        }
    }    
    return (float)correct_num / (float)test_data.num_of_data;
}

/*
 * Calculate the entropy gain = H(Y) - H(Y|X)
 * H(Y|X) needs parameters p(X<a), p(Y=0|X<a), p(Y=0|X>=a)
 * Assuming binary classification problem
 */
void get_gain(TreeNode* node, SplitPoint& split, int feature_id){
    int total_sum = node->data_size;
    dbg_ensures(total_sum > 0);
    float sum_class_0 = get_total_array(node->histogram_id, feature_id, NEG_LABEL);
    float sum_class_1 = get_total_array(node->histogram_id, feature_id, POS_LABEL);
    dbg_assert((sum_class_1 - node->num_pos_label) < EPS);
    float left_sum_class_0 = sum_array(node->histogram_id, feature_id, NEG_LABEL, split.feature_value);
    float right_sum_class_0 = sum_class_0 - left_sum_class_0;
    float left_sum_class_1 = sum_array(node->histogram_id, feature_id, POS_LABEL, split.feature_value);
    float right_sum_class_1 = sum_class_1 - left_sum_class_1;
    float left_sum = left_sum_class_0 + left_sum_class_1;
    float right_sum = right_sum_class_0 + right_sum_class_1;

    float px = (left_sum_class_0 + left_sum_class_1) / (1.0 * total_sum); // p(x<a)
    float py_x0 = (left_sum <= EPS) ? 0.f : left_sum_class_0 / left_sum;                            // p(y=0|x < a)
    float py_x1 = (right_sum <= EPS) ? 0.f : right_sum_class_0 / right_sum;                          // p(y=0|x >= a)
    // printf("sum_class_1=%f, sum_class_0=%f, right_sum = %f, right_sum_class_0 = %f right_sum_class_1= %f\n", sum_class_1, sum_class_0, right_sum, right_sum_class_0, right_sum_class_1);
    // printf("py_x0 = %f, py_x1 = %f\n", py_x0, py_x1);
    dbg_ensures(py_x0 >= -EPS && py_x0 <= 1+EPS);
    dbg_ensures(py_x1 >= -EPS && py_x1 <= 1+EPS);
    dbg_ensures(px >= -EPS && px <= 1+EPS);
    float entropy_left = ((1-py_x0) < EPS || py_x0 < EPS) ? 0 : -py_x0 * log2((double)py_x0) - (1-py_x0)*log2((double)1-py_x0);
    float entropy_right = ((1-py_x1) < EPS || py_x1 < EPS) ? 0 : -py_x1 * log2((double)py_x1) - (1-py_x1)*log2((double)1-py_x1);
    float H_YX = px * entropy_left + (1-px) * entropy_right;
    float px_prior = sum_class_0 / (sum_class_0 + sum_class_1);
    dbg_ensures(px_prior > 0 && px_prior < 1);
    split.entropy = ((1-px_prior) < EPS || px_prior < EPS) ? 0 : -px_prior * log2((double)px_prior) - (1-px_prior) * log2((double)1-px_prior);
    split.gain = split.entropy - H_YX;
    // printf("%f = %f - %f\n", split.gain, split.entropy, H_YX);
    dbg_ensures(split.gain >= -EPS);
}

/*
 * This function return the best split point at a given leaf node.
 * Best split is store in `split`
*/
void DecisionTree::find_best_split(TreeNode *node, SplitPoint &split)
{              
    assert(node != NULL);

    std::vector<SplitPoint> results;

    for (int i = 0; i < num_of_features; i++)
    {
        // merge different labels
        // put the result back into (node->histogram_id, i, 0)
        for (int k = 1; k < num_of_classes; k++) {
            merge_array(node->histogram_id, i, 0, node->histogram_id, i, k);
        }

        std::vector<float> possible_splits;
        uniform_array(possible_splits, node->histogram_id, i, 0);        
        dbg_assert(possible_splits.size() <= max_bin_size);
        for (auto& split_value: possible_splits)
        {
            SplitPoint t = SplitPoint(i, split_value, datasetPointer);
            get_gain(node, t, i);
            results.push_back(t);
        }
    }
    std::vector<SplitPoint>::iterator best_split = std::max_element(results.begin(), results.end(),
                                                                    [](const SplitPoint &l, const SplitPoint &r) { return l.gain < r.gain; });

    split.feature_id = best_split->feature_id;
    split.feature_value = best_split->feature_value;
    split.gain = best_split->gain;
}

/*
 * This function compress the data into histograms.
 * Each unlabeled leaf would have a (num_feature, num_class) histograms
 * This function takes the assumption that each leaf is re-initialized (we use a batch mode)
*/
void DecisionTree::compress(vector<TreeNode *> &unlabeled_leaf) {
    int block_num = this->datasetPointer->num_of_data;
    int thread_per_block = num_of_features; 
    long long number = (long long) max_num_leaves * num_of_features * num_of_classes * ((max_bin_size + 1) * 2 + 1);        
    
    hipMemcpy(cuda_histogram_ptr,
        histogram,
        sizeof(float) * number,
        hipMemcpyHostToDevice);    
    hipMemcpy(cuda_histogram_id_ptr,
        this->datasetPointer->histogram_id_ptr,
        sizeof(int) * this->datasetPointer->num_of_data,
        hipMemcpyHostToDevice); 
                                
    // https://stackoverflow.com/questions/31598021/cuda-cudamemcpy-struct-of-arrays
    // reference for moving objects from host to device in CUDA

    histogram_update_kernel<<<block_num, thread_per_block>>>(
        block_num,
        num_of_features,                              
        cuda_histogram_ptr, 
        cuda_label_ptr,
        cuda_value_ptr,
        cuda_histogram_id_ptr,
        num_of_features,
        num_of_classes,
        max_bin_size);  
    hipDeviceSynchronize();       
    
    hipMemcpy(histogram,
        cuda_histogram_ptr,
        sizeof(float) * number,
        hipMemcpyDeviceToHost);  
    
    float *histo = NULL;
    int bin_size = 0;

    for (int i = 0; i < num_of_features; i++) {
        for (int j = 0; j < num_of_classes; j++) {
            histo = get_histogram_array(0, i, j, histogram, num_of_features, num_of_classes, max_bin_size);
            bin_size = get_bin_size(histo);
            printf("[%d][%d]: bin_size %d\n", i, j, bin_size);
        }
    }    
    
}

/*
 * Serial version of training.
*/
void DecisionTree::train_on_batch(Dataset &train_data)
{       
    float pos_rate = ((float) train_data.num_pos_label) / train_data.num_of_data;
    dbg_assert(pos_rate > 0 && pos_rate < 1);
    root->num_pos_label = train_data.num_pos_label;
    root->entropy = - pos_rate * log2((double)pos_rate) - (1-pos_rate) * log2((double)(1-pos_rate));
    batch_initialize(root); // Reinitialize every leaf in T as unlabeled.
    vector<TreeNode *> unlabeled_leaf = __get_unlabeled(root);
    dbg_assert(unlabeled_leaf.size() <= max_num_leaves);
    while (!unlabeled_leaf.empty())
    {        
        // each while loop would add a new level node.
        this->cur_depth++;
        printf("depth [%d] finished\n", this->cur_depth);
        vector<TreeNode *> unlabeled_leaf_new; 
        if (unlabeled_leaf.size() > max_num_leaves) {
            for (int i = 0; i < unlabeled_leaf.size(); i++) {
                unlabeled_leaf[i]->set_label();
                this->num_leaves++;
            }
            break;
        }       
        init_histogram(unlabeled_leaf);
        Timer t1 = Timer();
        t1.reset();
        compress(unlabeled_leaf); 
        COMPRESS_TIME += t1.elapsed();         
        for (auto &cur_leaf : unlabeled_leaf)
        {            
            if (is_terminated(cur_leaf))
            {         
                cur_leaf->set_label();
                this->num_leaves++;             
            }
            else
            {                
                SplitPoint best_split;
                Timer t2 = Timer();
                t2.reset();
                find_best_split(cur_leaf, best_split);
                SPLIT_TIME += t2.elapsed();                
                dbg_ensures(best_split.gain >= -EPS);
                if (best_split.gain <= min_gain){
                    printf("Node terminated: gain=%.4f <= %.4f\n", min_node_size, best_split.gain, min_gain);
                    cur_leaf->set_label();
                    this->num_leaves++;               
                    continue;
                }
                cur_leaf->left_node = new TreeNode(this->cur_depth, this->num_nodes++, datasetPointer);
                cur_leaf->right_node = new TreeNode(this->cur_depth, this->num_nodes++, datasetPointer);
                cur_leaf->split(best_split, cur_leaf->left_node, cur_leaf->right_node);
                cur_leaf->is_leaf = false;
                cur_leaf->label = -1;
                unlabeled_leaf_new.push_back(cur_leaf->left_node);
                unlabeled_leaf_new.push_back(cur_leaf->right_node);
            }
        }
        unlabeled_leaf = unlabeled_leaf_new;
        unlabeled_leaf_new.clear(); 
    }
    self_check();    
}


void DecisionTree::self_check(){
    queue<TreeNode *> q;
    q.push(root);
    int count_leaf=0;
    int count_nodes=0;
    while (!q.empty())
    {
        auto tmp_ptr = q.front();
        q.pop();
        count_nodes++;
        if (tmp_ptr == NULL)
        {
            // should never reach here.
            fprintf(stderr, "ERROR: The tree contains node that have only one child\n");
            exit(-1);
        }
        else if ((tmp_ptr->left_node == NULL) && (tmp_ptr->right_node == NULL))
        {
            dbg_requires(tmp_ptr->is_leaf);
            dbg_requires(tmp_ptr->label == POS_LABEL || tmp_ptr->label == NEG_LABEL);
            count_leaf++;
        }
        else
        {
            dbg_requires(!tmp_ptr->is_leaf);
            dbg_requires(tmp_ptr->label == -1);
            q.push(tmp_ptr->left_node);
            q.push(tmp_ptr->right_node);
        }
    }
    dbg_assert(count_leaf == num_leaves);
    dbg_assert(count_nodes == num_nodes);
    printf("------------------------------------------------\n");
    printf("| Num_leaf: %d, num_nodes: %d, max_depth: %d | \n", num_leaves, num_nodes, cur_depth);
    printf("------------------------------------------------\n");

}


/* 
 * This function reture all the unlabeled leaf nodes in a breadth-first manner.
*/
vector<TreeNode *> DecisionTree::__get_unlabeled(TreeNode *node)
{
    queue<TreeNode *> q;
    q.push(node);
    vector<TreeNode *> ret;
    while (!q.empty())
    {
        auto tmp_ptr = q.front();
        q.pop();
        if (tmp_ptr == NULL)
        {
            // should never reach here.
            fprintf(stderr, "ERROR: The tree contains node that have only one child\n");
            exit(-1);
        }
        else if ((tmp_ptr->left_node == NULL) && (tmp_ptr->right_node == NULL) && tmp_ptr->label < 0)
        {
            ret.push_back(tmp_ptr);
        }
        else
        {
            q.push(tmp_ptr->left_node);
            q.push(tmp_ptr->right_node);
        }
    }
    return ret;
}

/*
 * initialize each leaf as unlabeled.
 */
void DecisionTree::batch_initialize(TreeNode *node)
{
    int feature_id = 0, class_id = 0;

    if (node == NULL)
    {
        // should never reach here.
        fprintf(stderr, "ERROR: The tree contains node that have only one child\n");
        exit(-1);
    }
    else if ((node->left_node == NULL) && (node->right_node == NULL))
    {
       node->init();
    }
    else
    {
        batch_initialize(node->left_node);
        batch_initialize(node->right_node);
    }
    return;
}


/*
 *
 */
TreeNode *DecisionTree::navigate(int data_index)
{
    TreeNode *ptr = this->root;
    while (!ptr->is_leaf)
    {
        dbg_assert(ptr->right_node != NULL && ptr->left_node != NULL);
        ptr = (ptr->split_ptr.decision_rule(data_index)) ? ptr->right_node : ptr->left_node;
    }
    return ptr;
}

/*
 * This function initialize the histogram for each unlabeled leaf node.
 * Also, potentially, it would free the previous histogram.
 */
void DecisionTree::init_histogram(vector<TreeNode *> &unlabeled_leaf)
{
    int c = 0;      
    assert(unlabeled_leaf.size() <= max_num_leaves);
    
    for (auto &p : unlabeled_leaf)
        p->histogram_id = c++;
}
